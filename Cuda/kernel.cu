
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <cstdio>


// Device input vectors
int *d_a;
//Device output vector
int *d_b;


__device__ int mod(int a, int b) {
    return a >= 0 ? a%b :  ( b - abs ( a%b ) ) % b;
}


__global__ void naivePrefixSum(int *A, int *B, int size, int iteration) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        if (index >= (1 << (iteration - 1)))
            A[index] = B[(int)(index - (1 << (iteration - 1)))] + B[index];
        else
            A[index] = B[index];
//        int aux = A[index];
//        A[index] = B[index];
//        B[index] = aux;
    }
}

void initCuda(int size) {
    // Allocate memory for each vector on GPU
    hipMalloc((void **) &d_a, size*sizeof(int));
    hipMalloc((void **) &d_b, size*sizeof(int));
}

void destroyCuda() {
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);

}

void runCuda(int *A, int size) {

    // Size, in bytes, of each vector
    size_t bytes = size*sizeof(int);


    // Copy host vectors to device
    hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, A, bytes, hipMemcpyHostToDevice);


//    int blockSize, gridSize, n;

//    // Tamaño de la matriz.
//    n = height*width;
//
//    // Tamaño del bloque. Elegir entre 32 y 31.
//    //blockSize = 32;
//    blockSize = 32;
//
//    // Number of thread blocks in grid
//    gridSize = (int)ceil((float)n/blockSize);
    int *aux;
    // Execute the kernel
    for (int i = 1; i <= (int)log2(size); ++i) {
        naivePrefixSum<<< size, 1 >>>(d_a, d_b, size, i);
        aux = d_b;
        d_b = d_a;
        d_a = aux;
    }

    // Copy array back to host
    hipMemcpy( A, d_b, bytes, hipMemcpyDeviceToHost );




}